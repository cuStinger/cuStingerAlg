#include "hip/hip_runtime.h"



#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <inttypes.h>

#include <math.h>

#include "update.hpp"
#include "cuStinger.hpp"

#include "operators.cuh"

#include "static_katz_centrality/katz.cuh"

#include "kernel_mergesort.hxx"
using namespace mgpu;


namespace cuStingerAlgs {

void katzCentrality::Init(cuStinger& custing){
	hostKatzData.nPathsCurr = (unsigned long long int*) allocDeviceArray(custing.nv+1, sizeof(unsigned long long int));
	hostKatzData.nPathsPrev = (unsigned long long int*) allocDeviceArray(custing.nv+1, sizeof(unsigned long long int));
	hostKatzData.vertexArray = (vertexId_t*) allocDeviceArray(custing.nv+1, sizeof(vertexId_t));
	hostKatzData.KC         = (double*) allocDeviceArray(custing.nv+1, sizeof(double));
	hostKatzData.lowerBound = (double*) allocDeviceArray(custing.nv+1, sizeof(double));
	hostKatzData.lowerBoundSort = (double*) allocDeviceArray(custing.nv+1, sizeof(double));
	hostKatzData.upperBound = (double*) allocDeviceArray(custing.nv+1, sizeof(double));

	deviceKatzData = (katzData*)allocDeviceArray(1, sizeof(katzData));

	cusLB = new cusLoadBalance(custing);

	SyncDeviceWithHost();
	Reset();
}

void katzCentrality::Reset(){
	hostKatzData.iteration = 1;

	SyncDeviceWithHost();
	copyArrayHostToDevice(&hostKatzData,deviceKatzData,1, sizeof(katzData));
}

void katzCentrality::setInputParameters(length_t K_,length_t maxDegree_, length_t maxIteration_){
	hostKatzData.K=K_;
	hostKatzData.maxDegree=maxDegree_;
	hostKatzData.maxIteration=maxIteration_;
	hostKatzData.alpha = 1.0/((double)hostKatzData.maxDegree+1.0);
}


void katzCentrality::Release(){
	// free(cusLB);
	delete cusLB;
	freeDeviceArray(deviceKatzData);
	freeDeviceArray(hostKatzData.nPathsCurr);
	freeDeviceArray(hostKatzData.nPathsPrev);
	freeDeviceArray(hostKatzData.vertexArray);
	freeDeviceArray(hostKatzData.KC);
	freeDeviceArray(hostKatzData.lowerBound);
	freeDeviceArray(hostKatzData.lowerBoundSort);
	freeDeviceArray(hostKatzData.upperBound);
}

void katzCentrality::Run(cuStinger& custing){

	allVinG_TraverseVertices<katzCentralityOperator::init>(custing,deviceKatzData);

	// GET MAX DEGREE
	standard_context_t context(false);

	hostKatzData.iteration = 1;
	
	hostKatzData.nActive = custing.nv;
	while(hostKatzData.nActive  > hostKatzData.K){

		hostKatzData.alphaI          = pow(hostKatzData.alpha,hostKatzData.iteration);
		hostKatzData.lowerBoundConst = pow(hostKatzData.alpha,hostKatzData.iteration+1)/((1.0-hostKatzData.alpha));
		hostKatzData.upperBoundConst = pow(hostKatzData.alpha,hostKatzData.iteration+1)/((1.0-hostKatzData.alpha*(double)hostKatzData.maxDegree));

		//cout << hostKatzData.iteration << " " << hostKatzData.alphaI << " " << hostKatzData.lowerBoundConst << " " << hostKatzData.upperBoundConst << endl;
		SyncDeviceWithHost();

		allVinG_TraverseVertices<katzCentralityOperator::initNumPathsPerIteration>(custing,deviceKatzData);
		allVinA_TraverseEdges_LB<katzCentralityOperator::updatePathCount>(custing,deviceKatzData,*cusLB);
		allVinG_TraverseVertices<katzCentralityOperator::updateKatzAndBounds>(custing,deviceKatzData);

		SyncHostWithDevice();
		hostKatzData.iteration++;

		unsigned long long int* temp = hostKatzData.nPathsCurr; hostKatzData.nPathsCurr=hostKatzData.nPathsPrev; hostKatzData.nPathsPrev=temp;
		// printf("%p %p %p\n",temp, hostKatzData.nPathsCurr,hostKatzData.nPathsPrev); 

		hostKatzData.nActive = 0;
		SyncDeviceWithHost();

		mergesort(hostKatzData.lowerBoundSort,hostKatzData.vertexArray,custing.nv, greater_t<double>(),context);

		// TODO I don't know when I need to sync the device with the host
		allVinG_TraverseVertices<katzCentralityOperator::countActive>(custing,deviceKatzData);
		//allVinA_TraverseVertices<katzCentralityOperator::printKID>(custing,deviceKatzData,hostKatzData.vertexArray, custing.nv);
/* 	unsigned long long int* nPathsCurr = (unsigned long long int*) allocHostArray(custing.nv+1, sizeof(unsigned long long int));
	unsigned long long int* nPathsPrev = (unsigned long long int*) allocHostArray(custing.nv+1, sizeof(unsigned long long int));
	vertexId_t* vertexArray = (vertexId_t*) allocHostArray(custing.nv+1, sizeof(vertexId_t));
	double* KC         = (double*) allocHostArray(custing.nv+1, sizeof(double));
	double* lowerBound = (double*) allocHostArray(custing.nv+1, sizeof(double));
	double* upperBound = (double*) allocHostArray(custing.nv+1, sizeof(double));
    
	copyArrayDeviceToHost(hostKatzData.lowerBound,lowerBound,custing.nv, sizeof(double)) ;
	copyArrayDeviceToHost(hostKatzData.upperBound,upperBound,custing.nv, sizeof(double)) ;
	copyArrayDeviceToHost(hostKatzData.KC,KC,custing.nv, sizeof(double)) ;
	copyArrayDeviceToHost(hostKatzData.vertexArray,vertexArray,custing.nv, sizeof(vertexId_t)) ;

//	for (int i=0; i<10; i++){
//	  printf("%d : katz = %g    lower = %g    upper=%g\n",vertexArray[i], KC[vertexArray[i]],lowerBound[vertexArray[i]],upperBound[vertexArray[i]]);
//	}

  	freeHostArray(nPathsCurr);
	freeHostArray(nPathsPrev);
    freeHostArray(vertexArray);
	freeHostArray(KC);
    freeHostArray(lowerBound);
	freeHostArray(upperBound);
*/		
		SyncHostWithDevice();
		cout << hostKatzData.nActive << endl;
	}
}

length_t katzCentrality::getIterationCount(){
	SyncHostWithDevice();
	return hostKatzData.iteration;
}


}// cuStingerAlgs namespace
