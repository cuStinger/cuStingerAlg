#include "hip/hip_runtime.h"



#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <inttypes.h>

#include <math.h>

#include "update.hpp"
#include "cuStinger.hpp"

#include "operators.cuh"

#include "static_katz_centrality/katz.cuh"

#include "kernel_mergesort.hxx"
using namespace mgpu;


namespace cuStingerAlgs {

void katzCentrality::Init(cuStinger& custing){
	hostKatzData.nPathsCurr = (length_t*) allocDeviceArray(custing.nv+1, sizeof(length_t));
	hostKatzData.nPathsPrev = (length_t*) allocDeviceArray(custing.nv+1, sizeof(length_t));
	hostKatzData.vertexArray = (vertexId_t*) allocDeviceArray(custing.nv+1, sizeof(vertexId_t));
	hostKatzData.KC         = (double*) allocDeviceArray(custing.nv+1, sizeof(double));
	hostKatzData.lowerBound = (double*) allocDeviceArray(custing.nv+1, sizeof(double));
	hostKatzData.upperBound = (double*) allocDeviceArray(custing.nv+1, sizeof(double));

	deviceKatzData = (katzData*)allocDeviceArray(1, sizeof(katzData));

	cusLB = new cusLoadBalance(custing);

	SyncDeviceWithHost();
	Reset();
}

void katzCentrality::Reset(){
	hostKatzData.iteration = 1;

	SyncDeviceWithHost();
	copyArrayHostToDevice(&hostKatzData,deviceKatzData,1, sizeof(katzData));
}


void katzCentrality::Release(){
	// free(cusLB);
	delete cusLB;
	freeDeviceArray(deviceKatzData);
	freeDeviceArray(hostKatzData.nPathsCurr);
	freeDeviceArray(hostKatzData.nPathsPrev);
	freeDeviceArray(hostKatzData.vertexArray);
	freeDeviceArray(hostKatzData.KC);
	freeDeviceArray(hostKatzData.lowerBound);
	freeDeviceArray(hostKatzData.upperBound);
}

void katzCentrality::Run(cuStinger& custing){

	allVinG_TraverseVertices<katzCentralityOperator::initNumPaths>(custing,deviceKatzData);

	// GET MAX DEGREE
	standard_context_t context(false);

	hostKatzData.iteration = 1;
	
	hostKatzData.nActive = custing.nv;
	while(hostKatzData.nActive  > hostKatzData.K ){

		hostKatzData.alphaI = pow(hostKatzData.alpha,hostKatzData.iteration);
		hostKatzData.upperBoundConst = pow(hostKatzData.alpha,hostKatzData.iteration+1)/((1.0-hostKatzData.alpha*(double)hostKatzData.maxDegree));
		hostKatzData.lowerBoundConst = pow(hostKatzData.alpha,hostKatzData.iteration+1)/((1.0-hostKatzData.alpha));
		SyncDeviceWithHost();

		allVinG_TraverseVertices<katzCentralityOperator::initNumPathsPerIteration>(custing,deviceKatzData);
		allVinA_TraverseEdges_LB<katzCentralityOperator::updatePathCount>(custing,deviceKatzData,*cusLB);
		allVinG_TraverseVertices<katzCentralityOperator::updateKatzAndBounds>(custing,deviceKatzData);

		SyncHostWithDevice();
		hostKatzData.iteration++;

		length_t* temp = hostKatzData.nPathsCurr; hostKatzData.nPathsCurr=hostKatzData.nPathsPrev; hostKatzData.nPathsPrev=temp;
		SyncDeviceWithHost();

		mergesort(hostKatzData.lowerBound,hostKatzData.vertexArray,custing.nv, less_t<double>(),context);

		// TODO I don't know when I need to sync the device with the host
		hostKatzData.nActive = 0;
		allVinG_TraverseVertices<katzCentralityOperator::countActive>(custing,deviceKatzData);

	}
}




}// cuStingerAlgs namespace
