#include "hip/hip_runtime.h"



#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <inttypes.h>

#include <math.h>

#include "update.hpp"
#include "cuStinger.hpp"

#include "operators.cuh"

#include "static_katz_centrality/katz.cuh"

#include "kernel_mergesort.hxx"
using namespace mgpu;


namespace cuStingerAlgs {

void katzCentrality::setInitParameters(length_t maxIteration_, bool isStatic_){
	hostKatzData.maxIteration=maxIteration_;
	isStatic = isStatic_;

	if(maxIteration_==0){
		cout << "Number of max iterations should be greater than zero" << endl;
		return;
	}
}


void katzCentrality::Init(cuStinger& custing){

	hostKatzData.nv = custing.nv;

	if(isStatic==true){
		hostKatzData.nPathsData = (ulong_t*) allocDeviceArray(2*(hostKatzData.nv), sizeof(ulong_t));
		hostKatzData.nPathsPrev = hostKatzData.nPathsData;
		hostKatzData.nPathsCurr = hostKatzData.nPathsData+(hostKatzData.nv);
	}
	else{
		hostKatzData.nPathsData = (ulong_t*) allocDeviceArray((hostKatzData.nv)*hostKatzData.maxIteration, sizeof(ulong_t));
		hostKatzData.nPaths = (ulong_t**) allocDeviceArray(hostKatzData.maxIteration, sizeof(ulong_t*));

		// Allocating 
		hPathsPtr = (ulong_t**)allocHostArray(hostKatzData.maxIteration, sizeof(ulong_t*));
		for(int i=0; i< hostKatzData.maxIteration; i++){
			hPathsPtr[i] = (hostKatzData.nPathsData+(hostKatzData.nv)*i);
		}
		hostKatzData.nPathsPrev = hPathsPtr[0];
		hostKatzData.nPathsCurr = hPathsPtr[1];

		copyArrayHostToDevice(hPathsPtr,hostKatzData.nPaths,hostKatzData.maxIteration,sizeof(double));
	}

	hostKatzData.vertexArray = (vertexId_t*) allocDeviceArray(hostKatzData.nv, sizeof(vertexId_t));
	hostKatzData.KC         = (double*) allocDeviceArray(hostKatzData.nv, sizeof(double));
	hostKatzData.lowerBound = (double*) allocDeviceArray(hostKatzData.nv, sizeof(double));
	hostKatzData.lowerBoundSort = (double*) allocDeviceArray(hostKatzData.nv, sizeof(double));
	hostKatzData.upperBound = (double*) allocDeviceArray(hostKatzData.nv, sizeof(double));

	deviceKatzData = (katzData*)allocDeviceArray(1, sizeof(katzData));
	cusLB = new cusLoadBalance(custing);

	SyncDeviceWithHost();
	Reset();
}

void katzCentrality::Reset(){
	hostKatzData.iteration = 1;

	if(isStatic==true){
		hostKatzData.nPathsPrev = hostKatzData.nPathsData;
		hostKatzData.nPathsCurr = hostKatzData.nPathsData+(hostKatzData.nv);
	}
	else{
		hostKatzData.nPathsPrev = hPathsPtr[0];
		hostKatzData.nPathsCurr = hPathsPtr[1];
	}

	SyncDeviceWithHost();
	copyArrayHostToDevice(&hostKatzData,deviceKatzData,1, sizeof(katzData));
}

void katzCentrality::setInputParameters(length_t K_,length_t maxDegree_){
	hostKatzData.K=K_;
	hostKatzData.maxDegree=maxDegree_;
	hostKatzData.alpha = 1.0/((double)hostKatzData.maxDegree+1.0);
}


void katzCentrality::Release(){
	delete cusLB;
	freeDeviceArray(deviceKatzData);
	freeDeviceArray(hostKatzData.nPathsData);

	if (!isStatic){
		freeDeviceArray(hostKatzData.nPaths);
		freeHostArray(hPathsPtr);
	}

	freeDeviceArray(hostKatzData.vertexArray);
	freeDeviceArray(hostKatzData.KC);
	freeDeviceArray(hostKatzData.lowerBound);
	freeDeviceArray(hostKatzData.lowerBoundSort);
	freeDeviceArray(hostKatzData.upperBound);
}

void katzCentrality::Run(cuStinger& custing){

	allVinG_TraverseVertices<katzCentralityOperator::init>(custing,deviceKatzData);
	// allVinG_TraverseVertices<katzCentralityOperator::printKID>(custing,deviceKatzData);
	// printf("\n");
	standard_context_t context(false);

	hostKatzData.iteration = 1;
	
	hostKatzData.nActive = hostKatzData.nv;
	while(hostKatzData.nActive  > hostKatzData.K && hostKatzData.iteration < hostKatzData.maxIteration){

		hostKatzData.alphaI          = pow(hostKatzData.alpha,hostKatzData.iteration);
		hostKatzData.lowerBoundConst = pow(hostKatzData.alpha,hostKatzData.iteration+1)/((1.0-hostKatzData.alpha));
		hostKatzData.upperBoundConst = pow(hostKatzData.alpha,hostKatzData.iteration+1)/((1.0-hostKatzData.alpha*(double)hostKatzData.maxDegree));

		//cout << hostKatzData.iteration << " " << hostKatzData.alphaI << " " << hostKatzData.lowerBoundConst << " " << hostKatzData.upperBoundConst << endl;
		SyncDeviceWithHost();

		allVinG_TraverseVertices<katzCentralityOperator::initNumPathsPerIteration>(custing,deviceKatzData);
		allVinA_TraverseEdges_LB<katzCentralityOperator::updatePathCount>(custing,deviceKatzData,*cusLB);
		allVinG_TraverseVertices<katzCentralityOperator::updateKatzAndBounds>(custing,deviceKatzData);

		SyncHostWithDevice();
		hostKatzData.iteration++;

		if(isStatic){
			// printf("^\n");
			// Swapping pointers.
			ulong_t* temp = hostKatzData.nPathsCurr; hostKatzData.nPathsCurr=hostKatzData.nPathsPrev; hostKatzData.nPathsPrev=temp;	
		// printf("prev  - %p\n ", hostKatzData.nPathsPrev);
		// printf("curr  - %p\n ", hostKatzData.nPathsCurr);
		// return;
			// copyArrayDeviceToDevice(hostKatzData.nPathsCurr,hostKatzData.nPathsPrev,hostKatzData.nv, sizeof(ulong_t));
		}else{
			// printf("@\n");
			hostKatzData.nPathsPrev = hPathsPtr[hostKatzData.iteration - 1];
			hostKatzData.nPathsCurr = hPathsPtr[hostKatzData.iteration - 0];
		}
		// printf("prev  - %p\n ", hostKatzData.nPathsPrev);
		// printf("curr  - %p\n ", hostKatzData.nPathsCurr);


		hostKatzData.nActive = 0;
		SyncDeviceWithHost();

		mergesort(hostKatzData.lowerBoundSort,hostKatzData.vertexArray,custing.nv, greater_t<double>(),context);

		allVinG_TraverseVertices<katzCentralityOperator::countActive>(custing,deviceKatzData);
		//allVinA_TraverseVertices<katzCentralityOperator::printKID>(custing,deviceKatzData,hostKatzData.vertexArray, custing.nv);


/* 	ulong_t* nPathsCurr = (ulong_t*) allocHostArray(hostKatzData.nv, sizeof(ulong_t));
	ulong_t* nPathsPrev = (ulong_t*) allocHostArray(hostKatzData.nv, sizeof(ulong_t));
	vertexId_t* vertexArray = (vertexId_t*) allocHostArray(hostKatzData.nv, sizeof(vertexId_t));
	double* KC         = (double*) allocHostArray(hostKatzData.nv, sizeof(double));
	double* lowerBound = (double*) allocHostArray(hostKatzData.nv, sizeof(double));
	double* upperBound = (double*) allocHostArray(hostKatzData.nv, sizeof(double));
    
	copyArrayDeviceToHost(hostKatzData.lowerBound,lowerBound,custing.nv, sizeof(double)) ;
	copyArrayDeviceToHost(hostKatzData.upperBound,upperBound,custing.nv, sizeof(double)) ;
	copyArrayDeviceToHost(hostKatzData.KC,KC,custing.nv, sizeof(double)) ;
	copyArrayDeviceToHost(hostKatzData.vertexArray,vertexArray,custing.nv, sizeof(vertexId_t)) ;

//	for (int i=0; i<10; i++){
//	  printf("%d : katz = %g    lower = %g    upper=%g\n",vertexArray[i], KC[vertexArray[i]],lowerBound[vertexArray[i]],upperBound[vertexArray[i]]);
//	}

  	freeHostArray(nPathsCurr);
	freeHostArray(nPathsPrev);
    freeHostArray(vertexArray);
	freeHostArray(KC);
    freeHostArray(lowerBound);
	freeHostArray(upperBound);
*/		
		SyncHostWithDevice();
		cout << hostKatzData.nActive << endl;
	}
}

length_t katzCentrality::getIterationCount(){
	SyncHostWithDevice();
	return hostKatzData.iteration;
}


}// cuStingerAlgs namespace
