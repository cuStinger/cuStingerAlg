#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <inttypes.h>

#include "static_triangle_counting/cct.hpp"
 
__device__ void conditionalWarpReduceIP(volatile triangle_t* sharedData,int blockSize,int dataLength){
  if(blockSize >= dataLength){
    if(threadIdx.x < (dataLength/2))
    {sharedData[threadIdx.x] += sharedData[threadIdx.x+(dataLength/2)];}
    __syncthreads();
  }
}

__device__ void warpReduceIP(triangle_t* __restrict__ outDataPtr,
    volatile triangle_t* __restrict__ sharedData,int blockSize){
  conditionalWarpReduceIP(sharedData,blockSize,64);
  conditionalWarpReduceIP(sharedData,blockSize,32);
  conditionalWarpReduceIP(sharedData,blockSize,16);
  conditionalWarpReduceIP(sharedData,blockSize,8);
  conditionalWarpReduceIP(sharedData,blockSize,4);
  if(threadIdx.x == 0)
    {*outDataPtr= sharedData[0] + sharedData[1];}
  __syncthreads();
}

__device__ void conditionalReduceIP(volatile triangle_t* __restrict__ sharedData,int blockSize,int dataLength){
	if(blockSize >= dataLength){
		if(threadIdx.x < (dataLength/2))
		{sharedData[threadIdx.x] += sharedData[threadIdx.x+(dataLength/2)];}
		__syncthreads();
	}
	if((blockSize < dataLength) && (blockSize > (dataLength/2))){
		if(threadIdx.x+(dataLength/2) < blockSize){
			sharedData[threadIdx.x] += sharedData[threadIdx.x+(dataLength/2)];
		}
		__syncthreads();
	}
}

__device__ void blockReduceIP(triangle_t* __restrict__ outGlobalDataPtr,
    volatile triangle_t* __restrict__ sharedData,int blockSize){
  __syncthreads();
  conditionalReduceIP(sharedData,blockSize,1024);
  conditionalReduceIP(sharedData,blockSize,512);
  conditionalReduceIP(sharedData,blockSize,256);
  conditionalReduceIP(sharedData,blockSize,128);

  warpReduceIP(outGlobalDataPtr, sharedData, blockSize);
  __syncthreads();
}

__device__ void initializeIP(const vertexId_t diag_id, const length_t u_len, length_t v_len,
    length_t* const __restrict__ u_min, length_t* const __restrict__ u_max,
    length_t* const __restrict__ v_min, length_t* const __restrict__ v_max,
    int* const __restrict__ found)
{
	if (diag_id == 0){
		*u_min=*u_max=*v_min=*v_max=0;
		*found=1;
	}
	else if (diag_id < u_len){
		*u_min=0; *u_max=diag_id;
		*v_max=diag_id;*v_min=0;
	}
	else if (diag_id < v_len){
		*u_min=0; *u_max=u_len;
		*v_max=diag_id;*v_min=diag_id-u_len;
	}
	else{
		*u_min=diag_id-v_len; *u_max=u_len;
		*v_min=diag_id-u_len; *v_max=v_len;
	}
}

__device__ void workPerThreadIP(const length_t uLength, const length_t vLength, 
	const int threadsPerIntersection, const int threadId,
    int * const __restrict__ outWorkPerThread, int * const __restrict__ outDiagonalId){
  int totalWork = uLength + vLength;
  int remainderWork = totalWork%threadsPerIntersection;
  int workPerThread = totalWork/threadsPerIntersection;

  int longDiagonals  = (threadId > remainderWork) ? remainderWork:threadId;
  int shortDiagonals = (threadId > remainderWork) ? (threadId - remainderWork):0;

  *outDiagonalId = ((workPerThread+1)*longDiagonals) + (workPerThread*shortDiagonals);
  *outWorkPerThread = workPerThread + (threadId < remainderWork);
}

__device__ void bSearchIP(unsigned int found, const vertexId_t diagonalId,
    vertexId_t const * const __restrict__ uNodes, vertexId_t const * const __restrict__ vNodes,
    length_t const * const __restrict__ uLength, 
    length_t * const __restrict__ outUMin, length_t * const __restrict__ outUMax,
    length_t * const __restrict__ outVMin, length_t * const __restrict__ outVMax,    
    length_t * const __restrict__ outUCurr,
    length_t * const __restrict__ outVCurr){
  	length_t length;
	
	while(!found) {
	    *outUCurr = (*outUMin + *outUMax)>>1;
	    *outVCurr = diagonalId - *outUCurr;
	    if(*outVCurr >= *outVMax){
			length = *outUMax - *outUMin;
			if(length == 1){
				found = 1;
				continue;
			}
	    }

	    unsigned int comp1 = uNodes[*outUCurr] > vNodes[*outVCurr-1];
	    unsigned int comp2 = uNodes[*outUCurr-1] > vNodes[*outVCurr];
	    if(comp1 && !comp2){
			found = 1;
	    }
	    else if(comp1){
	      *outVMin = *outVCurr;
	      *outUMax = *outUCurr;
	    }
	    else{
	      *outVMax = *outVCurr;
	      *outUMin = *outUCurr;
	    }
  	}

	if((*outVCurr >= *outVMax) && (length == 1) && (*outVCurr > 0) &&
	(*outUCurr > 0) && (*outUCurr < (*uLength - 1))){
		unsigned int comp1 = uNodes[*outUCurr] > vNodes[*outVCurr - 1];
		unsigned int comp2 = uNodes[*outUCurr - 1] > vNodes[*outVCurr];
		if(!comp1 && !comp2){(*outUCurr)++; (*outVCurr)--;}
	}
}

__device__ int fixStartPointIP(const length_t uLength, const length_t vLength,
    length_t * const __restrict__ uCurr, length_t * const __restrict__ vCurr,
    vertexId_t const * const __restrict__ uNodes, vertexId_t const * const __restrict__ vNodes){
	
	unsigned int uBigger = (*uCurr > 0) && (*vCurr < vLength) && (uNodes[*uCurr-1] == vNodes[*vCurr]);
	unsigned int vBigger = (*vCurr > 0) && (*uCurr < uLength) && (vNodes[*vCurr-1] == uNodes[*uCurr]);
	*uCurr += vBigger;
	*vCurr += uBigger;
	return (uBigger + vBigger);
}

__device__ void intersectPath(const length_t uLength, const length_t vLength,
    vertexId_t const * const __restrict__ uNodes, vertexId_t const * const __restrict__ vNodes,
    length_t * const __restrict__ uCurr, length_t * const __restrict__ vCurr,
    int * const __restrict__ workIndex, int * const __restrict__ workPerThread,
    int * const __restrict__ triangles, int found)
{
  if((*uCurr < uLength) && (*vCurr < vLength)){
    int comp;
    while(*workIndex < *workPerThread){
		comp = uNodes[*uCurr] - vNodes[*vCurr];
		*triangles += (comp == 0);
		*uCurr += (comp <= 0);
		*vCurr += (comp >= 0);
		*workIndex += (comp == 0) + 1;

		if((*vCurr == vLength) || (*uCurr == uLength)){
			break;
		}
    }
    *triangles -= ((comp == 0) && (*workIndex > *workPerThread) && (found));
  }
}


// u_len < v_len
__device__ triangle_t singleIntersection(vertexId_t u, vertexId_t const * const __restrict__ u_nodes, length_t u_len,
    vertexId_t v, vertexId_t const * const __restrict__ v_nodes, length_t v_len, int threads_per_block,
    volatile vertexId_t* __restrict__ firstFound, int tId)
{
	// Partitioning the work to the multiple thread of a single GPU processor. The threads should get a near equal number of the elements to Tersect - this number will be off by 1.
	int work_per_thread, diag_id;
	workPerThreadIP(u_len, v_len, threads_per_block, tId, &work_per_thread, &diag_id);
	triangle_t triangles = 0;
	int work_index = 0,found=0;
	length_t u_min,u_max,v_min,v_max,u_curr,v_curr;

	firstFound[tId]=0;

	if(work_per_thread>0){
		// For the binary search, we are figuring out the initial poT of search.
		initializeIP(diag_id, u_len, v_len,&u_min, &u_max,&v_min, &v_max,&found);
    	u_curr = 0; v_curr = 0;
	    bSearchIP(found, diag_id, u_nodes, v_nodes, &u_len, &u_min, &u_max, &v_min,
        &v_max, &u_curr, &v_curr);

    	int sum = fixStartPointIP(u_len, v_len, &u_curr, &v_curr, u_nodes, v_nodes);
    	work_index += sum;
	    if(tId > 0)
	      firstFound[tId-1] = sum;
	    triangles += sum;
	    intersectPath(u_len, v_len, u_nodes, v_nodes, &u_curr, &v_curr,
	        &work_index, &work_per_thread, &triangles, firstFound[tId]);
	}
	return triangles;
}

__device__ void workPerBlockIP(const vertexId_t numVertices,
    vertexId_t * const __restrict__ outMpStart,
    vertexId_t * const __restrict__ outMpEnd, int blockSize)
{
	vertexId_t verticesPerMp = numVertices/gridDim.x;
	vertexId_t remainderBlocks = numVertices % gridDim.x;
	vertexId_t extraVertexBlocks = (blockIdx.x > remainderBlocks)? remainderBlocks:blockIdx.x;
	vertexId_t regularVertexBlocks = (blockIdx.x > remainderBlocks)? blockIdx.x - remainderBlocks:0;

	vertexId_t mpStart = ((verticesPerMp+1)*extraVertexBlocks) + (verticesPerMp*regularVertexBlocks);
	*outMpStart = mpStart;
	*outMpEnd = mpStart + verticesPerMp + (blockIdx.x < remainderBlocks);
}


__global__ void devicecuStingerAllTriangles(cuStinger* custing,
    triangle_t * const __restrict__ outPutTriangles, const int threads_per_block,
    const int number_blocks, const int shifter)
{
	vertexId_t nv = custing->nv;
	// Partitioning the work to the multiple thread of a single GPU processor. The threads should get a near equal number of the elements to intersect - this number will be off by no more than one.
	int tx = threadIdx.x;
 	vertexId_t this_mp_start, this_mp_stop;

	const int blockSize = blockDim.x;
	workPerBlockIP(nv, &this_mp_start, &this_mp_stop, blockSize);

	__shared__ triangle_t  s_triangles[1024];
	__shared__ vertexId_t firstFound[1024];

	length_t adj_offset=tx>>shifter;
	length_t* firstFoundPos=firstFound + (adj_offset<<shifter);
	for (vertexId_t src = this_mp_start; src < this_mp_stop; src++){
		// int srcLen=d_off[src+1]-d_off[src];
		length_t srcLen=custing->dVD->getUsed()[src];
	    triangle_t tCount = 0;	    
		// for(int iter=d_off[src]+adj_offset; iter<d_off[src+1]; iter+=number_blocks){
		for(int k=adj_offset; k<srcLen; k+=number_blocks){
			// int dest = d_ind[k];
			vertexId_t dest = custing->dVD->getAdj()[src]->dst[k];
			// int destLen = d_off[dest+1]-d_off[dest];
			int destLen=custing->dVD->getUsed()[dest];

			// if (dest<src) 
			// 	continue;

			bool avoidCalc = (src == dest) || (destLen < 2) || (srcLen < 2);
			if(avoidCalc)
				continue;

	        bool sourceSmaller = (srcLen<destLen);
	        vertexId_t small = sourceSmaller? src : dest;
	        vertexId_t large = sourceSmaller? dest : src;
	        length_t small_len = sourceSmaller? srcLen : destLen;
	        length_t large_len = sourceSmaller? destLen : srcLen;


	        // int const * const small_ptr = d_ind + d_off[small];
	        // int const * const large_ptr = d_ind + d_off[large];
	        const vertexId_t* small_ptr = custing->dVD->getAdj()[small]->dst;
	        const vertexId_t* large_ptr = custing->dVD->getAdj()[large]->dst;
	        tCount += singleIntersection(small, small_ptr, small_len,
						large,large_ptr, large_len,
						threads_per_block,firstFoundPos,
						tx%threads_per_block);
		}
		s_triangles[tx] = tCount;
		blockReduceIP(&outPutTriangles[src],s_triangles,blockSize);
	}
}

void callDeviceAllTriangles(cuStinger& custing,
    triangle_t * const __restrict__ outPutTriangles, const int threads_per_block,
    const int number_blocks, const int shifter, const int thread_blocks, const int blockdim){

	devicecuStingerAllTriangles<<<thread_blocks, blockdim>>>(custing.devicePtr(), outPutTriangles, threads_per_block,number_blocks,shifter);
}





