

	
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <inttypes.h>

#include <iomanip> 

#include <cub.cuh>
#include <util_allocator.cuh>

#include <device/device_reduce.cuh>
#include <kernel_mergesort.hxx>


#include "update.hpp"
#include "cuStinger.hpp"

#include "operators.cuh"

#include "static_page_rank/pr.cuh"


using namespace hipcub;
using namespace mgpu;

CachingDeviceAllocator  g_allocator(true);  // Caching allocator for device memory

namespace cuStingerAlgs {

void StaticPageRank::Init(cuStinger& custing){
	hostPRData.nv = custing.nv;
	hostPRData.prevPR  = (prType*) allocDeviceArray(hostPRData.nv+1, sizeof(prType));
	hostPRData.currPR  = (prType*) allocDeviceArray(hostPRData.nv+1, sizeof(prType));
	hostPRData.absDiff = (prType*) allocDeviceArray(hostPRData.nv+1, sizeof(prType));
	hostPRData.contri = (prType*) allocDeviceArray(hostPRData.nv+1, sizeof(prType));

	hostPRData.reductionOut = (prType*) allocDeviceArray(1, sizeof(prType));
	// hostPRData.reduction=NULL;

	devicePRData = (pageRankData*)allocDeviceArray(1, sizeof(pageRankData));
	SyncDeviceWithHost();

	cusLB = new cusLoadBalance(custing,false,true);


	Reset();
}

void StaticPageRank::Reset(){
	hostPRData.iteration = 0;

	SyncDeviceWithHost();
}


void StaticPageRank::Release(){
	free(cusLB);	
	freeDeviceArray(devicePRData);
	freeDeviceArray(hostPRData.currPR);
	freeDeviceArray(hostPRData.prevPR);
	freeDeviceArray(hostPRData.absDiff);
	// freeDeviceArray(hostPRData.reduction);
	freeDeviceArray(hostPRData.reductionOut);
	freeDeviceArray(hostPRData.contri);
}

void StaticPageRank::Run(cuStinger& custing){

	allVinG_TraverseVertices<StaticPageRankOperator::init>(custing,devicePRData);
	hostPRData.iteration = 0;

	prType h_out = hostPRData.threshhold+1;

	while(hostPRData.iteration < hostPRData.iterationMax && h_out>hostPRData.threshhold){
		SyncDeviceWithHost();

		allVinA_TraverseVertices<StaticPageRankOperator::resetCurr>(custing,devicePRData,*cusLB);
		allVinA_TraverseVertices<StaticPageRankOperator::computeContribuitionPerVertex>(custing,devicePRData,*cusLB);
		allVinA_TraverseEdges_LB<StaticPageRankOperator::addContribuitionsUndirected>(custing,devicePRData,*cusLB);
		// allVinA_TraverseEdges_LB<StaticPageRankOperator::addContribuitions>(custing,devicePRData,*cusLB);
		allVinA_TraverseVertices<StaticPageRankOperator::dampAndDiffAndCopy>(custing,devicePRData,*cusLB);

		allVinG_TraverseVertices<StaticPageRankOperator::sum>(custing,devicePRData);
		SyncHostWithDevice();

		copyArrayDeviceToHost(hostPRData.reductionOut,&h_out, 1, sizeof(prType));

		hostPRData.iteration++;
	}
}

void StaticPageRank::setInputParameters(length_t prmIterationMax, prType prmThreshhold,prType prmDamp){
	hostPRData.iterationMax=prmIterationMax;
	hostPRData.threshhold=prmThreshhold;
	hostPRData.damp=prmDamp;
	hostPRData.normalizedDamp=(1-hostPRData.damp)/float(hostPRData.nv);
	SyncDeviceWithHost();
}

length_t StaticPageRank::getIterationCount(){
	return hostPRData.iteration;
}

void StaticPageRank::printRankings(cuStinger& custing){

	prType* d_scores = (prType*)allocDeviceArray(hostPRData.nv, sizeof(prType));
	vertexId_t* d_ids = (vertexId_t*)allocDeviceArray(hostPRData.nv, sizeof(vertexId_t));

	copyArrayDeviceToDevice(hostPRData.currPR, d_scores,hostPRData.nv, sizeof(prType));


	allVinG_TraverseVertices<StaticPageRankOperator::setIds>(custing,d_ids);

	standard_context_t context(false);
	mergesort(d_scores,d_ids,hostPRData.nv,greater_t<float>(),context);

	prType* h_scores = (prType*)allocHostArray(hostPRData.nv, sizeof(prType));
	vertexId_t* h_ids    = (vertexId_t*)allocHostArray(hostPRData.nv, sizeof(vertexId_t));

	copyArrayDeviceToHost(d_scores,h_scores,hostPRData.nv, sizeof(prType));
	copyArrayDeviceToHost(d_ids,h_ids,hostPRData.nv, sizeof(vertexId_t));

	for(int v=0; v<10; v++){
		printf("Pr[%d]:= %f\n",h_ids[v],h_scores[v]);
	}

	allVinG_TraverseVertices<StaticPageRankOperator::resetCurr>(custing,devicePRData);
	allVinG_TraverseVertices<StaticPageRankOperator::sumPr>(custing,devicePRData);

		// SyncHostWithDevice();
	prType h_out;

		copyArrayDeviceToHost(hostPRData.reductionOut,&h_out, 1, sizeof(prType));
		cout << "                     " << setprecision(9) << h_out << endl;


	freeDeviceArray(d_scores);
	freeDeviceArray(d_ids);
	freeHostArray(h_scores);
	freeHostArray(h_ids);
}

}// cuStingerAlgs namespace






