#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <inttypes.h>

#include <math.h>


#include "utils.hpp"
#include "update.hpp"
#include "cuStinger.hpp"

#include "algs.cuh"

#include "static_katz_centrality/katz.cuh"
#include "streaming_katz_centrality/katz.cuh"

using namespace cuStingerAlgs;


#define CUDA(call, ...) do {                        \
        hipError_t _e = (call);                    \
        if (_e == hipSuccess) break;               \
        fprintf(stdout,                             \
                "CUDA runtime error: %s (%d)\n",    \
                hipGetErrorString(_e), _e);        \
        return -1;                                  \
    } while (0)


void generateEdgeUpdates(length_t nv, length_t numEdges, vertexId_t* edgeSrc, vertexId_t* edgeDst){
	for(int32_t e=0; e<numEdges; e++){
		edgeSrc[e] = rand()%nv;
		edgeDst[e] = rand()%nv;
	}
}



int main(const int argc, char *argv[]){
	int device=0;
    hipSetDevice(device);
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, device);
 
    length_t nv, ne,*off;
    vertexId_t *adj;

	bool isDimacs,isSNAP,isRmat=false,isMarket;
	string filename(argv[1]);
	isDimacs = filename.find(".graph")==std::string::npos?false:true;
	isSNAP   = filename.find(".txt")==std::string::npos?false:true;
	isRmat 	 = filename.find("kron")==std::string::npos?false:true;
	isMarket = filename.find(".mtx")==std::string::npos?false:true;

	if(isDimacs){
	    readGraphDIMACS(argv[1],&off,&adj,&nv,&ne,isRmat);
	}
	else if(isSNAP){
	    readGraphSNAP(argv[1],&off,&adj,&nv,&ne,isRmat);
	}
	else if(isMarket){
		readGraphMatrixMarket(argv[1],&off,&adj,&nv,&ne,(isRmat)?false:true);
	}
	else{ 
		cout << "Unknown graph type" << endl;
	}

	cout << "Vertices: " << nv << "    Edges: " << ne << endl;

	hipEvent_t ce_start,ce_stop;
	cuStinger custing(defaultInitAllocater,defaultUpdateAllocater);

	cuStingerInitConfig hipInit;
	hipInit.initState =eInitStateCSR;
	hipInit.maxNV = nv+1;
	hipInit.useVWeight = false;
	hipInit.isSemantic = false;  // Use edge types and vertex types
	hipInit.useEWeight = false;
	// CSR data
	hipInit.csrNV 			= nv;
	hipInit.csrNE	   		= ne;
	hipInit.csrOff 			= off;
	hipInit.csrAdj 			= adj;
	hipInit.csrVW 			= NULL;
	hipInit.csrEW			= NULL;

	custing.initializeCuStinger(hipInit);

	
	float totalTime;

	// Finding largest vertex
	vertexId_t maxV=0;
	length_t   maxLen=0;
	for(int v=1; v<nv;v++){
		if((off[v+1]-off[v])>maxLen){
			maxV=v;
			maxLen=off[v+1]-off[v];
		}
	}
	katzCentrality kc;
	kc.setInitParameters(20,100,maxLen,true);
	kc.Init(custing);
	kc.Reset();
	start_clock(ce_start, ce_stop);
	kc.Run(custing);
	totalTime = end_clock(ce_start, ce_stop);
	cout << "The number of iterations      : " << kc.getIterationCount() << endl;
	cout << "Total time for KC             : " << totalTime << endl; 
	cout << "Average time per iteartion    : " << totalTime/(float)kc.getIterationCount() << endl; 
	kc.Release();

	katzCentralityStreaming kcs;

	kcs.setInitParameters(20,100,maxLen);
	kcs.Init(custing);
	start_clock(ce_start, ce_stop);
	kcs.runStatic(custing);
	totalTime = end_clock(ce_start, ce_stop);
	cout << "The number of iterations      : " << kcs.getIterationCount() << endl;
	cout << "Total time for KC             : " << totalTime << endl; 
	cout << "Average time per iteartion    : " << totalTime/(float)kcs.getIterationCount() << endl; 

	int numBatchEdges=1;

	BatchUpdateData bud(numBatchEdges,true);

	generateEdgeUpdates(nv, numBatchEdges, bud.getSrc(),bud.getDst());

	// BatchUpdate bu(bud);
	BatchUpdate* bu = new BatchUpdate(bud);

	start_clock(ce_start, ce_stop);
	kcs.insertedBatchUpdate(custing,*bu);
	totalTime = end_clock(ce_start, ce_stop);


	// cout << "The number of iterations      : " << kcs.getIterationCount() << endl;
	cout << "Total time for KC streaming   : " << totalTime << endl; 
	// cout << "Average time per iteartion    : " << totalTime/(float)kcs.getIterationCount() << endl; 


	katzCentrality kcPostUpdate;
	kcPostUpdate.setInitParameters(20,100,maxLen,true);
	kcPostUpdate.Init(custing);
	kcPostUpdate.Reset();
	start_clock(ce_start, ce_stop);
	kcPostUpdate.Run(custing);
	totalTime = end_clock(ce_start, ce_stop);
	cout << "The number of iterations      : " << kcPostUpdate.getIterationCount() << endl;
	cout << "Total time for KC             : " << totalTime << endl; 
	cout << "Average time per iteartion    : " << totalTime/(float)kcPostUpdate.getIterationCount() << endl; 


	double* kcScoresStreaming  = (double*) allocHostArray(custing.nv, sizeof(double));
	double* kcScoresPostUpdate = (double*) allocHostArray(custing.nv, sizeof(double));

	kcs.copyKCToHost(kcScoresStreaming);
	kcPostUpdate.copyKCToHost(kcScoresPostUpdate);

	for(int i=0; i < 100; i++){
		// printf("%1.11lf, ", kcScoresStreaming[i]-kcScoresPostUpdate[i]);
	}
	printf("\n");

	double sum=0.0;
	for(int i=0; i < custing.nv; i++){
		sum += fabs(kcScoresStreaming[i]-kcScoresPostUpdate[i]);
	}
	printf("Sum of difference %4.11lf \n", sum);


	kcPostUpdate.Release();
	kcs.Release();

	custing.freecuStinger();

	free(off);
	free(adj);

	hipDeviceReset();
    return 0;	
}

