#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <inttypes.h>

#include <math.h>


#include "utils.hpp"
#include "update.hpp"
#include "cuStinger.hpp"

#include "algs.cuh"

#include "static_katz_centrality/katz.cuh"
#include "streaming_katz_centrality/katz.cuh"

using namespace cuStingerAlgs;


#define CUDA(call, ...) do {                        \
        hipError_t _e = (call);                    \
        if (_e == hipSuccess) break;               \
        fprintf(stdout,                             \
                "CUDA runtime error: %s (%d)\n",    \
                hipGetErrorString(_e), _e);        \
        return -1;                                  \
    } while (0)


void generateEdgeUpdates(length_t nv, length_t numEdges, vertexId_t* edgeSrc, vertexId_t* edgeDst){
	for(int32_t e=0; e<numEdges; e++){
		edgeSrc[e] = rand()%nv;
		edgeDst[e] = rand()%nv;
	}
}



int main(const int argc, char *argv[]){
	int device=0;
    hipSetDevice(device);
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, device);
 
    length_t nv, ne,*off;
    vertexId_t *adj;

	bool isDimacs,isSNAP,isRmat=false,isMarket;
	string filename(argv[1]);
	isDimacs = filename.find(".graph")==std::string::npos?false:true;
	isSNAP   = filename.find(".txt")==std::string::npos?false:true;
	isRmat 	 = filename.find("kron")==std::string::npos?false:true;
	isMarket = filename.find(".mtx")==std::string::npos?false:true;

	if(isDimacs){
	    readGraphDIMACS(argv[1],&off,&adj,&nv,&ne,isRmat);
	}
	else if(isSNAP){
	    readGraphSNAP(argv[1],&off,&adj,&nv,&ne,isRmat);
	}
	else if(isMarket){
		readGraphMatrixMarket(argv[1],&off,&adj,&nv,&ne,(isRmat)?false:true);
	}
	else{ 
		cout << "Unknown graph type" << endl;
	}

	cout << "Vertices: " << nv << "    Edges: " << ne << endl;

	hipEvent_t ce_start,ce_stop;
	cuStinger custing(defaultInitAllocater,defaultUpdateAllocater);

	cuStingerInitConfig hipInit;
	hipInit.initState =eInitStateCSR;
	hipInit.maxNV = nv+1;
	hipInit.useVWeight = false;
	hipInit.isSemantic = false;  // Use edge types and vertex types
	hipInit.useEWeight = false;
	// CSR data
	hipInit.csrNV 			= nv;
	hipInit.csrNE	   		= ne;
	hipInit.csrOff 			= off;
	hipInit.csrAdj 			= adj;
	hipInit.csrVW 			= NULL;
	hipInit.csrEW			= NULL;

	custing.initializeCuStinger(hipInit);

	
	float totalTime;

	// Finding largest vertex
	vertexId_t maxV=0;
	length_t   maxLen=0;
	for(int v=1; v<nv;v++){
		if((off[v+1]-off[v])>maxLen){
			maxV=v;
			maxLen=off[v+1]-off[v];
		}
	}
	for (int r=0; r<1; r++){
		katzCentrality kc;
		kc.setInitParameters(20,100,maxLen,true);
		kc.Init(custing);
		kc.Reset();
		start_clock(ce_start, ce_stop);
		kc.Run(custing);
		totalTime = end_clock(ce_start, ce_stop);
		cout << "The number of iterations      : " << kc.getIterationCount() << endl;
		cout << "Total time for KC             : " << totalTime << endl; 
		cout << "Average time per iteartion    : " << totalTime/(float)kc.getIterationCount() << endl; 
		kc.Release();
	}
	for (int r=0; r<1; r++){

		katzCentrality kc2;
		kc2.setInitParameters(20,100,maxLen,false);
		kc2.Init(custing);
		kc2.Reset();
		start_clock(ce_start, ce_stop);
		kc2.Run(custing);
		totalTime = end_clock(ce_start, ce_stop);
		cout << "The number of iterations      : " << kc2.getIterationCount() << endl;
		cout << "Total time for KC             : " << totalTime << endl; 
		cout << "Average time per iteartion    : " << totalTime/(float)kc2.getIterationCount() << endl; 

		kc2.Release();
	}

	katzCentralityStreaming kcs;

	kcs.setInitParameters(20,100,maxLen);
	kcs.Init(custing);
	start_clock(ce_start, ce_stop);
	kcs.runStatic(custing);
	totalTime = end_clock(ce_start, ce_stop);
	cout << "The number of iterations      : " << kcs.getIterationCount() << endl;
	cout << "Total time for KC             : " << totalTime << endl; 
	cout << "Average time per iteartion    : " << totalTime/(float)kcs.getIterationCount() << endl; 

	int numBatchEdges=5000;

	BatchUpdateData bud(numBatchEdges,true);

	generateEdgeUpdates(nv, numBatchEdges, bud.getSrc(),bud.getDst());

	// BatchUpdate bu(bud);
	BatchUpdate* bu = new BatchUpdate(bud);

	start_clock(ce_start, ce_stop);
	kcs.insertedBatchUpdate(custing,*bu);
	totalTime = end_clock(ce_start, ce_stop);


	delete bu;

	// cout << "The number of iterations      : " << kcs.getIterationCount() << endl;
	cout << "Total time for KC streaming   : " << totalTime << endl; 
	// cout << "Average time per iteartion    : " << totalTime/(float)kcs.getIterationCount() << endl; 


	kcs.Release();

	custing.freecuStinger();

	free(off);
	free(adj);

	hipDeviceReset();
    return 0;	
}

